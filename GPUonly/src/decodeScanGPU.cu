#include "hip/hip_runtime.h"
#include<stdio.h>
#include<string.h>
#include<time.h>

#include<format.h>
#include<decodeScanGPU.h>
#include<entropyRLEdecodeGPU.h>



__host__ static void unstuffBuf(JPGReader *jpg) {
  clock_t start_t = clock();
  unsigned char next_val, *src, *dst, *buf = jpg->pos;
  for (src=buf, dst=buf; src < jpg->end;) {
    if((*dst++ = *src++) != 0xFF) continue;
    // Remove byte stuffing //
    if ((next_val = *src++) == 0x00) continue;
    // But keep restart markers //
    if (0xD0 == (next_val & 0xF8)) *dst++ = next_val;
  }
  jpg->end = dst;
  // Put an EOF marker at the new end
  *dst++ = 0xFF;
  *dst++ = 0xD9;
  clock_t end_t = clock();
  jpg->time += end_t - start_t;
}


// This only shows the bits, but doesn't move past them //
__host__ static int showBits(JPGReader* jpg, int num_bits) {
  if(!num_bits) return 0;

  while (jpg->num_bufbits < num_bits){
    unsigned char newbyte = (jpg->pos >= jpg->end) ? 0xFF : *jpg->pos++;
    jpg->bufbits = (jpg->bufbits << 8) | newbyte;
    jpg->num_bufbits += 8;
  }
  return (jpg->bufbits >> (jpg->num_bufbits - num_bits)) & ((1 << num_bits) - 1);
}


// Show the bits AND move past them //
__host__ static int getBits(JPGReader* jpg, int num_bits) {
  int res = showBits(jpg, num_bits);
  jpg->num_bufbits -= num_bits;
  return res;
}


__host__ static int getVLC(JPGReader* jpg, DhtVlc* vlc_table, unsigned char* code) {
  int symbol = showBits(jpg, 16);
  DhtVlc vlc = vlc_table[symbol];
  if(!vlc.num_bits) {
    jpg->error = SYNTAX_ERROR;
    return 0;
  }
  jpg->num_bufbits -= vlc.num_bits;  
  if(code) *code = vlc.tuple;
  unsigned char num_bits = vlc.tuple & 0x0F;
  if (!num_bits) return 0;
  int value = getBits(jpg, num_bits);
  if (value < (1 << (num_bits - 1))) {
    value += ((-1) << num_bits) + 1;
    /*short tmp = value;
    tmp += ((-1) << num_bits) + 1;
    value = tmp;*/
  }
  return value;  
}


__host__ static void decodeBlock(JPGReader* jpg, ColourChannel* channel) {
  unsigned char code = 0;
  int value, coef = 0;
  int* block = channel->working_space_pos;

  // Read DC value //
  channel->dc_cumulative_val += getVLC(jpg, &jpg->vlc_tables[channel->dc_id][0], NULL);
  block[0] = (channel->dc_cumulative_val) * jpg->dq_tables[channel->dq_id][0];
  // Read  AC values //
  do {
    value = getVLC(jpg, &jpg->vlc_tables[channel->ac_id][0], &code);
    if (!code) break; // EOB marker //
    if (!(code & 0x0F) && (code != 0xF0)) THROW(SYNTAX_ERROR);
    coef += (code >> 4) + 1;
    if (coef > 63) THROW(SYNTAX_ERROR);
    block[(int)deZigZag[coef]] = value * jpg->dq_tables[channel->dq_id][coef];
  } while(coef < 63);

  channel->working_space_pos += 64;
}


__host__ void decodeScanGPU(JPGReader* jpg) {
  unsigned char *pos = jpg->pos;
  unsigned int header_len = read16(pos);
  if (pos + header_len > jpg->end) THROW(SYNTAX_ERROR);
  pos += 2;
  
  // Read segment header //
  if (header_len < (4 + 2 * jpg->num_channels)) THROW(SYNTAX_ERROR);
  if (*(pos++) != jpg->num_channels) THROW(UNSUPPORTED_ERROR);
  int i;
  ColourChannel *channel;
  for(i = 0, channel=jpg->channels; i<jpg->num_channels; i++, channel++, pos+=2){
    if (pos[0] != channel->id) THROW(SYNTAX_ERROR);
    if (pos[1] & 0xEE) THROW(SYNTAX_ERROR);
    channel->dc_id = pos[1] >> 4;
    channel->ac_id = (pos[1] & 1) | 2;
  }
  if (pos[0] || (pos[1] != 63) || pos[2]) THROW(UNSUPPORTED_ERROR);
  pos = jpg->pos = jpg->pos + header_len;

  // Remove byte stuffing //
  unstuffBuf(jpg);

  // Do the decode scan //
  int restart_interval = jpg->restart_interval;
  if (!restart_interval) {
    
    /*jpg->device_pos = jpg->device_file_buf.mem + (jpg->pos - jpg->buf);
    int num_threads = (jpg->end - jpg->pos) * 8; // One thread per bit-position
    int threads_per_block = 256;
    int num_blocks = (num_threads + threads_per_block - 1) / threads_per_block;
    huffmanDecode_kernel<<<num_blocks, threads_per_block>>>(jpg->device_pos,
								num_threads,
								jpg->device_buf_values.mem,
								jpg->device_jump_lengths.mem,
								jpg->device_run_lengths.mem,
								jpg->device_vlc_tables);
								if (hipGetLastError() != hipSuccess) THROW(CUDA_KERNEL_LAUNCH_ERROR);*/
    
    
    for (int block_y = 0; block_y < jpg->num_blocks_y; block_y++){
      for (int block_x = 0; block_x < jpg->num_blocks_x; block_x++){
	// Loop over all channels //
	for (i = 0, channel = jpg->channels; i < jpg->num_channels; i++, channel++){
	  // Loop over samples in block //
	  for (int sample_y = 0; sample_y < channel->samples_y; ++sample_y){
	    for (int sample_x = 0; sample_x < channel->samples_x; ++sample_x){
	      decodeBlock(jpg, channel);
	      if (jpg->error) return;
	    }}}}}
    
  } else {

    int restart_count = restart_interval;
    int next_restart_index = 0;
    
    // Loop over all blocks
    for (int block_y = 0; block_y < jpg->num_blocks_y; block_y++){
      for (int block_x = 0; block_x < jpg->num_blocks_x; block_x++){

	// Loop over all channels //
	for (i = 0, channel = jpg->channels; i < jpg->num_channels; i++, channel++){

	  // Loop over samples in block //
	  for (int sample_y = 0; sample_y < channel->samples_y; ++sample_y){
	    for (int sample_x = 0; sample_x < channel->samples_x; ++sample_x){
	      decodeBlock(jpg, channel);
	      if (jpg->error) return;
	    }
	  }
	}

	if (restart_interval && !(--restart_count) && (jpg->pos < jpg->end)){
	  // Byte align //
	  jpg->num_bufbits &= 0xF8;
	  i = getBits(jpg, 16);
	  if (((i & 0xFFF8) != 0xFFD0) || ((i & 7) != next_restart_index))
	    THROW(SYNTAX_ERROR);
	  next_restart_index = (next_restart_index + 1) & 7;
	  restart_count = restart_interval;
	  for (i = 0; i < 3; i++)
	    jpg->channels[i].dc_cumulative_val = 0;
	}
      }
    }
  }
}
